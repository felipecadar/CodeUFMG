#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"

extern "C"{
    #include "structs.h"
}

__global__ void kernel() {
    printf("Hello, from the GPU! \n");
}

extern "C" void myfunction_GPU(Vec3 a, Vec3 b, Vec3 c) {

    printf("Cuda -> %.2f\n", a.dot(b));

    kernel<<<1,1>>>();
    hipDeviceSynchronize();
}